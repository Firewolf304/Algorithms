#include "hip/hip_runtime.h"
#include "../include/local.hpp"


auto generate_points = [](int count) {
    srand(static_cast<unsigned int>(time(0)));
    std::vector<Point> points;
    std::ofstream file("points.txt", std::ios::out | std::ios::trunc);
    for (int i = 0; i < count; ++i) {
        float x = static_cast<float>(rand() % 10000);
        float y = static_cast<float>(rand() % 10000);
        auto point = Point{x, y};
        points.emplace_back( point );
        file << point.x << " " << point.y<< std::endl;
    }
    file.close();
    return points;
};

auto read_file = [] (const std::string & filename) {
    std::vector<Point> points;
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Ошибка при открытии файла!" << std::endl;
        return points;
    }
    std::string line;
    while (std::getline(file, line)) {
        double x, y;
        std::istringstream iss(line);
        if (iss >> x >> y) {
            points.emplace_back(x, y);
        } else {
            std::cerr << "Ошибка при чтении строки: " << line << std::endl;
        }
    }
    file.close();
    return points;
};

int main() {

    /*std::vector<Point> points = {
            {1.0, 2.0}, {1.2, 2.1}, {0.9, 1.8}, {1.1, 2.2},  // Cluster 1
            {6.0, 8.0}, {5.8, 7.9}, {6.2, 8.1}, {5.9, 8.2},  // Cluster 2
            {10.0, 10.0}, {10.2, 10.3}, {9.8, 9.7}, {10.1, 10.1},  // Cluster 3
            {15.0, 2.0}, {14.8, 2.1}, {15.2, 1.9}, {15.1, 2.2},  // Cluster 4
            {20.0, 20.0}, {20.2, 20.3}, {19.9, 19.8}, {20.1, 20.1}  // Cluster 5
    };*/
    //auto points = generate_points(count);
    auto points = read_file("points.txt");
    auto centroids = read_file("centroids.txt");
    int count = points.size();
    int k = centroids.size();
    bool tests = true;
    kmeans::kmean_cpu cpu {points};
    kmeans::kmean_gpu gpu {points};
    //cpu.init_centroids(k);



    /*std::ofstream file("centroids.txt", std::ios::out | std::ios::trunc);
    for (auto i = 0; i < centroids.size(); i++) {
        file << "(" << centroids[i].x << " " << centroids[i].y << ") -> " << i << std::endl;
    }
    file.close();*/
    /*std::cout <<  ("+============ Centroids ============+\n");
    for (auto i = 0; i < centroids.size(); i++) {
        std::printf("(%f,%f) -> %d\n", centroids[i].x, centroids[i].y, i);
    }*/
    std::cout <<  ("+============ CPU ============+\n");

    cpu.start(k, centroids);
    cpu.output_sort_file();
    if(tests) {
        std::cout << ("TESTS:\n");
        std::cout << std::fixed << std::setprecision(20) << "Cophenetic correlation="
                  << cpu.calculateCopheneticCorrelation() << std::endl;
        std::cout << std::fixed << std::setprecision(20) << "Cophenetic error=" << cpu.estimateCopheneticError(30)
                  << std::endl;
    }


    std::cout << ("+============ GPU ============+\n");

    gpu.start(k, centroids);
    gpu.output_sort_file();
    //gpu.output_sort();
    if(tests) {
        std::cout << ("TESTS:\n");
        std::cout << std::fixed << std::setprecision(20) << "Cophenetic correlation="
                  << gpu.calculateCopheneticCorrelation() << std::endl;
        std::cout << std::fixed << std::setprecision(20) << "Cophenetic error=" << gpu.estimateCopheneticError(30)
                  << std::endl;
    }


    return 0;
}
