#include "hip/hip_runtime.h"
//
// Created by firewolf on 05.12.24.
//

#ifndef K_MEANS_GPU_CU
#define K_MEANS_GPU_CU

#include "libs.hpp"
#include "points/point.hxx"
namespace kmeans {
    class kmean_gpu;
     __global__ inline void updateCentroids(Point *points, int nPoints, int cluster, float *summX, float *summY, int *count) {
         extern __shared__ float sharedMemory[];
         float *localSumX = &sharedMemory[0];
         float *localSumY = &sharedMemory[blockDim.x];
         int *localCount = (int *)&sharedMemory[2 * blockDim.x];

         int tid = threadIdx.x + blockIdx.x * blockDim.x;
         int localIdx = threadIdx.x;

         // init local summm
         localSumX[localIdx] = 0;
         localSumY[localIdx] = 0;
         localCount[localIdx] = 0;

         // updates
         if (tid < nPoints) {
             Point p = points[tid];
             if (p.cluster == cluster) {
                 localSumX[localIdx] += p.x;
                 localSumY[localIdx] += p.y;
                 localCount[localIdx] += 1;
             }
         }
         __syncthreads();

         // Редуцируем
         for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
             if (localIdx < offset) {
                 // magic swap
                 localSumX[localIdx] += localSumX[localIdx + offset];
                 localSumY[localIdx] += localSumY[localIdx + offset];
                 localCount[localIdx] += localCount[localIdx + offset];
             }
             __syncthreads();
         }

         // update summs
         if (localIdx == 0) {
             atomicAdd(summX, localSumX[0]);
             atomicAdd(summY, localSumY[0]);
             atomicAdd(count, localCount[0]);
         }
    }
    __global__ inline void kernel_clusterPoints(Point * raw_points, Point * raw_centroids, int * changed, int nPoints, int k, int test = 0) {
        extern __shared__ Point sharedCentroids[];
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        if (threadIdx.x < k) {
            sharedCentroids[threadIdx.x] = raw_centroids[threadIdx.x];
        }

        __syncthreads();
        if(tid < nPoints) {
            Point &p = raw_points[tid];
            int nearestCluster = -1;
            float minDist = MAXFLOAT;
            for (int i = 0; i < k; ++i) {
                float dist = (p.x - sharedCentroids[i].x) * (p.x - sharedCentroids[i].x) +
                             (p.y - sharedCentroids[i].y) * (p.y - sharedCentroids[i].y);
                if (dist < minDist) {
                    minDist = dist;
                    nearestCluster = i;
                }
            }
            if (p.cluster != nearestCluster) {
                p.cluster = nearestCluster;
                atomicAdd(changed, 1);
            }
            if (test) {
                printf("tid=%d thread=%d block=%d dim=%d = (%f, %f) cluster=%d changed=%d\n", tid, threadIdx.x, blockIdx.x, blockDim.x, p.x, p.y, p.cluster, *changed);
            }
        }
    }

    using vars_test = std::variant<std::string, int, double, float>;
    using var_test = std::variant<vars_test, std::vector<vars_test>>;
    class kmean_gpu {
        thrust::device_vector<Point> points;
        thrust::device_vector<Point> centroids;
        int k = -1;
        bool test;
    public:
        kmean_gpu(const std::vector<Point> &points, bool test = false) :
                points(points.begin(), points.end()), test(test) {}
        __device__ __host__ double distance (Point & a, Point & b) {
            return std::sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
        }
        __device__ __host__ double distance (Point a, Point b) {
            return std::sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
        }
        __device__ __host__ double distance_sqrt (Point & a, Point & b) {
            return std::sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
        }
        Point calculateCentroid(int cluster) {
            /*float sumX = 0, sumY = 0;
            int count = 0;
            thrust::host_vector<Point> hostCounts = this->points;
            for (auto point : hostCounts) {
                if (point.cluster == cluster) {
                    sumX += point.x;
                    sumY += point.y;
                    ++count;
                    if(this->test) {
                        printf("Included point: (x=%f, y=%f) from cluster %d in centroid calculation.\n", point.x,
                               point.y, cluster);
                    }
                }
            }
            if(this->test) {
                printf("Cluster %d: Total Sum (X=%f, Y=%f), Count=%d.\n", cluster, sumX, sumY, count);
            }
            return count > 0 ? Point(sumX / count, sumY / count) : Point(0, 0);*/
            int nPoints = this->points.size();
            float *d_sumX, *d_sumY;
            int *d_count;
            hipMalloc(&d_sumX, sizeof(float));
            hipMalloc(&d_sumY, sizeof(float));
            hipMalloc(&d_count, sizeof(int));
            hipMemset(d_sumX, 0, sizeof(float));
            hipMemset(d_sumY, 0, sizeof(float));
            hipMemset(d_count, 0, sizeof(int));

            int threadsPerBlock = 256;
            int blocks = (nPoints + threadsPerBlock - 1) / threadsPerBlock;
            size_t sharedMemorySize = 2 * threadsPerBlock * sizeof(float) + threadsPerBlock * sizeof(int);

            updateCentroids<<<blocks, threadsPerBlock, sharedMemorySize>>>(
                    thrust::raw_pointer_cast(this->points.data()),
                    nPoints,
                    cluster,
                    d_sumX,
                    d_sumY,
                    d_count
            );

            float h_sumX, h_sumY;
            int h_count;
            hipMemcpy(&h_sumX, d_sumX, sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&h_sumY, d_sumY, sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

            hipFree(d_sumX);
            hipFree(d_sumY);
            hipFree(d_count);

            if (this->test) {
                printf("Cluster %d: Total Sum (X=%f, Y=%f), Count=%d.\n", cluster, h_sumX, h_sumY, h_count);
            }

            return h_count > 0 ? Point(h_sumX / h_count, h_sumY / h_count) : Point(0, 0);
        }
        // Старт подготовленных центроидов
        void start(int k, std::vector<Point> & init_centroids, bool output = true) {
            this->k = k;
            this->centroids = thrust::device_vector<Point>(init_centroids.begin(), init_centroids.end());
            start(output);
        }
        thrust::device_vector<Point> init_centroids(int k) {
            this->k = k;
            thrust::host_vector<Point> p = this->points;
            std::vector<Point> cent;
            std::random_device rd;
            std::mt19937 gen(rd());
            std::uniform_int_distribution<> distrib(0, this->points.size());
            for (int i = 0; i < k; ++i) {
                int idx = distrib(gen);
                cent.push_back(p[idx]);
            }
            p.clear();
            return {cent.begin(), cent.end()};
        }
        void start(bool output = true) {
            if(this->centroids.empty()) {
                throw std::runtime_error("Centroids are not initialized");
            }
            int nPoints = this->points.size();
            thrust::device_vector<int> clusterCounts(k);

            std::shared_ptr<int> changed = std::make_shared<int>(1);
            int count = 0;
            auto start = std::chrono::high_resolution_clock::now();

            int threadsPerBlock = 256;
            int blocks = (nPoints + threadsPerBlock - 1) / threadsPerBlock;

            while (*changed > 0) {
                if(this->test) fmt::println("=========== Iter {} ===========", count);
                *changed  = 0;
                // Не нарушаем warp-блоки


                kernel_clusterPoints<<<blocks, threadsPerBlock, this->k * sizeof(Point)>>>(
                        thrust::raw_pointer_cast((this->points.data())),
                        thrust::raw_pointer_cast((this->centroids.data())),
                        thrust::raw_pointer_cast(changed.get()),
                        nPoints,
                        this->k,
                        this->test
                        );

                thrust::fill(clusterCounts.begin(), clusterCounts.end(), 0);

                if(this->test) fmt::println("Centroids:");
                #pragma omp parallel for
                for (int i = 0; i < k; ++i) {
                    centroids[i] = calculateCentroid(i);
                }
                /*fmt::println("Centroids:", count);
                updateCentroids<<<blocks, threadsPerBlock>>>(
                        thrust::raw_pointer_cast(this->points.data()),
                        thrust::raw_pointer_cast(this->centroids.data()),
                        thrust::raw_pointer_cast(clusterCounts.data()),
                        nPoints, k
                );
                std::vector<int> hostCounts(k);
                thrust::copy(clusterCounts.begin(), clusterCounts.end(), hostCounts.begin());

                thrust::host_vector<Point> hostCentroids = this->centroids;
                for (int i = 0; i < k; ++i) {
                    if (hostCounts[i] > 0) {
                        hostCentroids[i].x /= hostCounts[i];
                        hostCentroids[i].y /= hostCounts[i];
                    }
                }
                this->centroids = hostCentroids;*/
                count++;
            }
            if(output) {
                auto end = std::chrono::high_resolution_clock::now();
                auto duration = end - start;
                std::cout << "Execution time: " << duration / std::chrono::milliseconds(1) << " ms" << std::endl;
            }
        }

        void output() {
            thrust::host_vector<Point> hostPoints = this->points;
            for(auto point : hostPoints) {
                fmt::print("({},{}) -> {}\n", point.x, point.y, point.cluster);
            }
        }
        void output_sort() {
            thrust::host_vector<Point> hostPoints = this->points;
            std::map<int, std::vector<Point>> mapper{};
            for(auto &point : hostPoints) {
                mapper[point.cluster].push_back(point);

            }
            for(auto cluster : mapper) {
                fmt::println("============= {} =============", cluster.first);
                for(auto & point : cluster.second) {
                    fmt::println("({} {}) -> {}", point.x, point.y, point.cluster);
                }
            }
        }

        void output_sort_file() {
            thrust::host_vector<Point> hostPoints = this->points;
            std::map<int, std::vector<Point>> mapper{};
            for(auto &point : hostPoints) {
                mapper[point.cluster].push_back(point);
            }
            std::ofstream file("output_gpu.txt", std::ios::out | std::ios::trunc);
            for(auto cluster : mapper) {
                //file << "============= " << cluster.first << " =============\n";
                for(auto & point : cluster.second) {
                    file << point.x << " " << point.y << " " << point.cluster << std::endl;
                }
            }
            file.close();
        }
        double calculateCopheneticCorrelation() {
            std::vector<double> dataDistances, clusterDistances;
            thrust::host_vector<Point> centroid = this->points;
            thrust::host_vector<Point> centroiders = this->centroids;
            for (size_t i = 0; i < centroid.size(); ++i) {
                for (size_t j = i + 1; j < centroid.size(); ++j) {
                    dataDistances.push_back(distance_sqrt(centroid[i], centroid[j]));
                }
            }

            for (size_t i = 0; i < centroid.size(); ++i) {
                for (size_t j = i + 1; j < centroid.size(); ++j) {
                    if (centroid[i].cluster == centroid[j].cluster) {
                        clusterDistances.push_back(0.0);
                    } else {
                        clusterDistances.push_back(distance_sqrt(centroiders[centroid[i].cluster], centroiders[centroid[j].cluster]));
                    }
                }
            }

            return pearsonCorrelation(dataDistances, clusterDistances);
        }

        double pearsonCorrelation(const std::vector<double>& x, const std::vector<double>& y) {
            double meanX = std::accumulate(x.begin(), x.end(), 0.0) / x.size();
            double meanY = std::accumulate(y.begin(), y.end(), 0.0) / y.size();

            double numerator = 0.0, denomX = 0.0, denomY = 0.0;

            for (size_t i = 0; i < x.size(); ++i) {
                double diffX = x[i] - meanX;
                double diffY = y[i] - meanY;
                numerator += diffX * diffY;
                denomX += diffX * diffX;
                denomY += diffY * diffY;
            }

            return numerator / std::sqrt(denomX * denomY);
        }
        double estimateCopheneticError(int numIterations) {
            std::vector<double> correlations;
            for (int i = 0; i < numIterations; ++i) {

                this->centroids = init_centroids(this->k);
                start(false);

                correlations.push_back(calculateCopheneticCorrelation());
            }
            double mean = std::accumulate(correlations.begin(), correlations.end(), 0.0) / correlations.size();
            double variance = 0.0;
            for (double corr : correlations) {
                variance += (corr - mean) * (corr - mean);
            }
            variance /= correlations.size();
            return std::sqrt(variance);
        }
    };
}

#endif //K_MEANS_GPU_CU
