#include "hip/hip_runtime.h"
//
// Created by firewolf on 05.12.24.
//

#ifndef K_MEANS_GPU_CU
#define K_MEANS_GPU_CU

#include "libs.hpp"
#include "points/point.hxx"
namespace kmeans {
    class kmean_gpu;
     __global__ inline void updateCentroids(Point* raw_points, Point* raw_centroids, int* clusterCounts, int nPoints, int k) {
        int tid = threadIdx.x + blockIdx.x * blockDim.x;

        if (tid < nPoints) {
            Point& p = raw_points[tid];
            atomicAdd(&(raw_centroids[p.cluster].x), p.x);
            atomicAdd(&(raw_centroids[p.cluster].y), p.y);
            atomicAdd(&(clusterCounts[p.cluster]), 1);
        }
    }
    __global__ inline void kernel_clusterPoints(Point * raw_points, Point * raw_centroids, int * changed, int nPoints, int k) {
        extern __shared__ Point sharedCentroids[];
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        if (threadIdx.x < k) {
            sharedCentroids[threadIdx.x] = raw_centroids[threadIdx.x];
        }
        __syncthreads();

        if(tid < nPoints) {
            Point& p = raw_points[tid];
            int nearestCluster = -1;
            float minDist = MAXFLOAT;
            for (int i = 0; i < k; ++i) {
                float dist = (p.x - sharedCentroids[i].x) * (p.x - sharedCentroids[i].x) +
                             (p.y - sharedCentroids[i].y) * (p.y - sharedCentroids[i].y);
                if (dist < minDist) {
                    minDist = dist;
                    nearestCluster = i;
                }
            }
            if (p.cluster != nearestCluster) {
                p.cluster = nearestCluster;
                atomicAdd(changed, 1);
            }
        }
    }

    using vars_test = std::variant<std::string, int, double, float>;
    using var_test = std::variant<vars_test, std::vector<vars_test>>;
    class kmean_gpu {
        thrust::device_vector<Point> points;
        thrust::device_vector<Point> centroids;
        int k = -1;
        bool test;
    public:
        kmean_gpu(const std::vector<Point> &points, bool test = false) :
                points(points.begin(), points.end()), test(test) {}
        __device__ __host__ double distance (Point & a, Point & b) {
            return std::sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
        }
        __device__ __host__ double distance (Point a, Point b) {
            return std::sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
        }
        __device__ __host__ double distance_sqrt (Point & a, Point & b) {
            return std::sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
        }

        // Старт подготовленных центроидов
        void start(int k, std::vector<Point> & init_centroids) {
            this->k = k;
            this->centroids = thrust::device_vector<Point>(init_centroids.begin(), init_centroids.end());
            start();
        }

        void start() {
            if(this->centroids.empty()) {
                throw std::runtime_error("Centroids are not initialized");
            }
            int nPoints = this->points.size();
            thrust::device_vector<int> clusterCounts(k);

            std::shared_ptr<int> changed;
            *changed = 1;
            while (*changed > 0) {
                *changed  = 0;
                // Не нарушаем warp-блоки
                int threadsPerBlock = 256;
                int blocks = (nPoints + threadsPerBlock - 1) / threadsPerBlock;
                kernel_clusterPoints<<<blocks, threadsPerBlock, k * sizeof(Point)>>>(
                        thrust::raw_pointer_cast((this->points.data())),
                        thrust::raw_pointer_cast((this->centroids.data())),
                        thrust::raw_pointer_cast(changed.get()),
                        nPoints,
                        this->k
                        );

                thrust::fill(clusterCounts.begin(), clusterCounts.end(), 0);
                updateCentroids<<<blocks, threadsPerBlock>>>(
                        thrust::raw_pointer_cast(this->points.data()),
                        thrust::raw_pointer_cast(this->centroids.data()),
                        thrust::raw_pointer_cast(clusterCounts.data()),
                        nPoints, k
                );
                std::vector<int> hostCounts(k);
                thrust::copy(clusterCounts.begin(), clusterCounts.end(), hostCounts.begin());

                thrust::host_vector<Point> hostCentroids = this->centroids;
                for (int i = 0; i < k; ++i) {
                    if (hostCounts[i] > 0) {
                        hostCentroids[i].x /= hostCounts[i];
                        hostCentroids[i].y /= hostCounts[i];
                    }
                }
                this->centroids = hostCentroids;

            }
        }

        void output() {
            thrust::host_vector<Point> hostPoints = this->points;
            for(auto point : hostPoints) {
                fmt::print("({},{}) -> {}\n", point.x, point.y, point.cluster);
            }
        }

        /*
         * Test values:
         *      distance - vector<double>
         */
        std::map<std::string, var_test> get_tests() {
            return {
                    {
                            "distance",
                            std::vector<vars_test>{
                                    distance( {0,0}, {1,1}),
                                    distance( {1,1}, {2,2}),
                                    distance( {-1,-1}, {2,2})
                            },
                    },
            };
        }
    };
}

#endif //K_MEANS_GPU_CU
